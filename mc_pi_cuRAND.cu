#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#define PI 3.14159265359

float calculate_accuracy(float pi);

__global__ void mc(float *d_x, float *d_y,int *d_area_count){
	int index = threadIdx.x + blockIdx.x*blockDim.x;
	if(d_x[index]*d_x[index] + d_y[index]*d_y[index] <= 1.0f){
		d_area_count[index] = 1;
	}
	else 
	{
		d_area_count[index] = 0;
	}
}

struct GpuTimer {
  hipEvent_t start;
  hipEvent_t stop;

  GpuTimer() {
    hipEventCreate(&start);
    hipEventCreate(&stop);
  }

  ~GpuTimer() {
    hipEventDestroy(start);
    hipEventDestroy(stop);
  }

  void Start() {
    hipEventRecord(start, 0);
  }

  void Stop() {
    hipEventRecord(stop, 0);
  }

  float Elapsed() {
    float elapsed;
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    return elapsed;
  }
};

int main(void){
	//Try different number of randoms used
	int num_randoms[] = {10,100,1000,10000,100000,1000000,10000000,100000000};
	for(int j = 0; j < sizeof(num_randoms)/sizeof(num_randoms[0]); j++)
	{
		//Initialize the timer to calculate the speed 
		GpuTimer timer;
		timer.Start();
		printf("Number of randoms used %d \n", num_randoms[j]);
		
		//Initialize variables
   		int N = num_randoms[j];
		int area=0;	
	
		//Allocate pointers for host and device memory
		float *h_x; float *h_y;
		float *d_x; float *d_y;
		int *d_area_count;
		int *h_area_count;

		int mem_size = N*sizeof(float);
		h_x = (float*)malloc(mem_size);
		h_y = (float*)malloc(mem_size);
		h_area_count = (int*)malloc(mem_size);
		hipMalloc((void**)&d_x,mem_size);
		hipMalloc((void**)&d_y,mem_size);
		hipMalloc((void**)&d_area_count,mem_size);
	
		//Declare variable
		hiprandGenerator_t gen;

		//Create random number generator
		hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);

		//Set the generator options
		hiprandSetPseudoRandomGeneratorSeed(gen,1234ULL);

		//Generate the randoms
		hiprandGenerateUniform(gen,d_x,N);
		hiprandGenerateUniform(gen,d_y,N);	
	
		mc<<<N/128,128>>>(d_x,d_y,d_area_count);

		//Copy result from device to host
		//cudaMemcpy(h_x,d_x,mem_size,cudaMemcpyDeviceToHost);
		//cudaMemcpy(h_y,d_y,mem_size,cudaMemcpyDeviceToHost);
	
		//printf("Success %f %f %f",h_x[0], h_x[1], h_x[2]);
		hipMemcpy(h_area_count,d_area_count,mem_size,hipMemcpyDeviceToHost);
		for(int i = 0; i < N; i++){
			area += h_area_count[i];
		}
		float pi = (4*area)/float(N);
		printf("Pi is %f \n",pi);
		printf("The error is %f \n", calculate_accuracy(pi));
		//Clean up memory
		free(h_x);
		free(h_y);
		hipFree(d_x);
		hipFree(d_y);

		//Stop the timer
		timer.Stop();
		printf("Execution time: %f\n", timer.Elapsed()/10.0);
	}	
	return(0);	
}

float calculate_accuracy(float pi){
	return float(PI - pi);
}

